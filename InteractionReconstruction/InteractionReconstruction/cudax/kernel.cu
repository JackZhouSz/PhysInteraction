#include "util/gl_wrapper.h" ///< for cuda_gl_interop
#include <cuda_gl_interop.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>

#include "cudax/kernel.h"
#include "cudax/CudaTimer.h"
#include "cudax/hip/hip_runtime_api.h" ///< SDK error checking
#include "cudax/CublasHelper.h"
#include "cudax/CudaHelper.h"
#include "cudax/KinectCamera.h"
#include "cudax/kernel_init.h"
#include "cudax/kernel_upload.h"
#include "cudax/kernel_debug.h"
#include "cudax/PixelIndexer.h"

#include "cudax/functors/IsSilhouette.h"
#include "cudax/functors/ComputeJacobianSilhouette.h"
//#include "cudax/functors/ComputeJacobianData.h"
#include "cudax/functors/ComputeJacobianDataPointCloud_ZH.h"
#include "cudax/functors/ComputeJacobianInteraction.h"

#include <iostream>
#include <fstream>
#include <string>
#include <iomanip> 

using namespace cudax;

struct absolute_value : public thrust::unary_function <float, float > {
	__host__ __device__
		float operator()(float x) const {
		return (x>=0) ? x : -x;
	}
};

void kernel_bind()
{
    if(cudax::sensor_depth)   CHECK_CUDA(hipBindTextureToArray(depth_tex, cudax::sensor_depth));
}

void kernel_unbind(){
    if(cudax::sensor_depth)   CHECK_CUDA(hipUnbindTexture(depth_tex));
    cudax::sensor_depth=NULL;
}

void kernel(float* eigen_JtJ, float* eigen_JtF, float & push_error, float & pull_error, bool eval_metric, bool reweight, int id, int iter, 
	int num_sensor_points, int num_rendered_points, int &push_num, int &pull_num) {
    
    // CUDA_TIMED_BLOCK(timer,"indexing constraints")

	int n_pull, n_push;

	if (settings->fit2D_enable) {
		n_push = 2 * num_rendered_points;
		pixel_indexer->clear_counters_memory();
		pixel_indexer->assign_pull_constraints_indices(num_sensor_points);
		n_pull = 1 * num_sensor_points; // point to plane
	}
	if (!settings->fit2D_enable) {
		pixel_indexer->clear_counters_memory();			
		pixel_indexer->assign_pull_constraints_indices(num_sensor_points);
		n_pull = 1 * num_sensor_points; // point to plane
		n_push = 0;
	}

	int n_total = n_pull + n_push;	
      
    // CUDA_TIMED_BLOCK(timer,"memory resize + zero (J+e)")
    { 
        const J_row zeros = {};
		thrust::fill(J->begin(), J->begin() + n_total, zeros);
		thrust::fill(F->begin(), F->begin() + n_total, 0.0f);
        
        if (n_total==0) return;
    }
    J_row* J_push = thrust::raw_pointer_cast(J->data());
    J_row* J_pull = J_push + n_push;    
    
	float* F_push = thrust::raw_pointer_cast(F->data());
    float* F_pull = F_push + n_push;

	ComputeJacobianSilhouette functor_push(J_push, F_push);
	ComputeJacobianData functor_data_model(J_pull, F_pull, reweight);

	int factor = 6;
    if(store_corresps) {
		hmodel_correspondences = new thrust::device_vector<float>(n_pull * factor, -111);
		functor_data_model.store_data(thrust::raw_pointer_cast(hmodel_correspondences->data()));
    }
    
	thrust::sequence(push_indices->begin(), push_indices->begin() + num_rendered_points);

    //CUDA_TIMED_BLOCK(timer,"Assemble Jacobian")
	{
		if (settings->fit2D_enable) {
			//thrust::for_each(_rendered_indicator->begin(), _rendered_indicator->begin() + num_rendered_points, functor_push);
			thrust::for_each(push_indices->begin(), push_indices->begin() + num_rendered_points, functor_push);
		}
		if (settings->fit3D_enable) 		
			thrust::for_each(_sensor_indicator->begin(), _sensor_indicator->begin() + num_sensor_points, functor_data_model);				
	} 

    // CUDA_TIMED_BLOCK(timer, "Jt*J and Jt*e + CPU Transfer")
	{
		CublasHelper::outer_product_J(*J, *JtJ, n_total, NUM_THETAS);
		CublasHelper::vector_product_J(*J, *F, *JtF, n_total, NUM_THETAS);
		thrust::copy(JtF->begin(), JtF->end(), eigen_JtF);
		thrust::copy(JtJ->begin(), JtJ->end(), eigen_JtJ);
	}

	// Multiply with CPU
	if (_test) {
		thrust::host_vector<float> F_host(n_total);
		thrust::copy(F->begin(), F->begin() + n_total, F_host.begin());
		thrust::host_vector<J_row> J_host(n_total);
		thrust::copy(J->begin(), J->begin() + n_total, J_host.begin());

		for (size_t i = 0; i < NUM_THETAS; i++) {
			double result = 0;
			for (size_t k = 0; k < n_total; k++) {				
				J_row a = J_host[k];
				result = result + (double)a.data[i] * (double)F_host[k];				
			}
			eigen_JtF[i] = result;
		}	
		for (size_t i = 0; i < NUM_THETAS; i++) {
			for (size_t j = 0; j < NUM_THETAS; j++) {
				double result = 0;
				for (size_t k = 0; k < n_total; k++) {
					J_row a = J_host[k];
					result = result + (double)a.data[i] * (double)a.data[j];
				}
				eigen_JtJ[i * NUM_THETAS + j] = result;
			}
		}
    }

    /// Only need evaluate metric on the last iteration
    if (eval_metric) {
		thrust::device_vector<float> f_pull(n_pull);
		thrust::transform(F->begin() + n_push, F->begin() + n_push + n_pull, f_pull.begin(), absolute_value());
		pull_error = thrust::reduce(f_pull.begin(), f_pull.end());
		pull_error = pull_error / n_pull;
		pull_num = n_pull;
		//std::cout << pull_error << std::endl;

		thrust::device_vector<float> f_push(n_push);
		thrust::transform(F->begin(), F->begin() + n_push, f_push.begin(), absolute_value());
		push_error = thrust::reduce(f_push.begin(), f_push.end());
		push_error = push_error / n_push;
		push_num = n_push;
	}		
	
	//Write the correspondences	
	if (store_corresps) {
		std::ofstream output_file;
		std::string data_path = "...";
		thrust::host_vector<float> output(n_pull * factor);
		
		thrust::copy(hmodel_correspondences->begin(), hmodel_correspondences->begin() + n_pull * factor, output.begin());
		output_file.open(data_path + "corresp-" + std::to_string(id) + ".txt");
		for (size_t i = 0; i < n_pull * factor; i++) {
			output_file << output[i] << " ";
		}
		output_file.close();
	}	
    return;
}


void kernel2(float* eigen_JtJ, float* eigen_JtF, float & push_error, float & pull_error, bool eval_metric, bool reweight, int id, int iter,
	int num_sensor_points, int num_rendered_points /*, int &push_num, int &pull_num*/, bool cal_conf) {

	// CUDA_TIMED_BLOCK(timer,"indexing constraints")

	int n_pull, n_push;

	if (settings->fit2D_enable) {
		n_push = 2 * num_rendered_points;
		pixel_indexer->clear_counters_memory();
		pixel_indexer->assign_pull_constraints_indices(num_sensor_points);
		n_pull = 1 * num_sensor_points; // point to plane
	}
	if (!settings->fit2D_enable) {
		pixel_indexer->clear_counters_memory();
		pixel_indexer->assign_pull_constraints_indices(num_sensor_points);
		n_pull = 1 * num_sensor_points; // point to plane
		n_push = 0;
	}

	int n_total = n_pull + n_push;

	// CUDA_TIMED_BLOCK(timer,"memory resize + zero (J+e)")
	{
		const J_row zeros = {};
		thrust::fill(J->begin(), J->begin() + n_total, zeros);
		thrust::fill(F->begin(), F->begin() + n_total, 0.0f);

		if (n_total == 0) return;
	}
	J_row* J_push = thrust::raw_pointer_cast(J->data());
	J_row* J_pull = J_push + n_push;

	float* F_push = thrust::raw_pointer_cast(F->data());
	float* F_pull = F_push + n_push;

	ComputeJacobianSilhouette_ZH functor_push(J_push, F_push);
	ComputeJacobianDataPointCloud functor_data_model(J_pull, F_pull, reweight, cal_conf);

	int factor = 6;
	if (store_corresps) {
		hmodel_correspondences = new thrust::device_vector<float>(n_pull * factor, -111);
		functor_data_model.store_data(thrust::raw_pointer_cast(hmodel_correspondences->data()));
	}

	thrust::sequence(push_indices->begin(), push_indices->begin() + num_rendered_points);

	//CUDA_TIMED_BLOCK(timer,"Assemble Jacobian")
	{
		if (settings->fit2D_enable) {
			//thrust::for_each(_rendered_indicator->begin(), _rendered_indicator->begin() + num_rendered_points, functor_push);
			thrust::for_each(push_indices->begin(), push_indices->begin() + num_rendered_points, functor_push);
		}
		if (settings->fit3D_enable)
			thrust::for_each(_point_index->begin(), _point_index->begin() + num_sensor_points, functor_data_model);
	}

	// CUDA_TIMED_BLOCK(timer, "Jt*J and Jt*e + CPU Transfer")
	{
		CublasHelper::outer_product_J(*J, *JtJ, n_total, NUM_THETAS);
		CublasHelper::vector_product_J(*J, *F, *JtF, n_total, NUM_THETAS);
		thrust::copy(JtF->begin(), JtF->end(), eigen_JtF);
		thrust::copy(JtJ->begin(), JtJ->end(), eigen_JtJ);
	}

	// Multiply with CPU
	if (_test) {
		thrust::host_vector<float> F_host(n_total);
		thrust::copy(F->begin(), F->begin() + n_total, F_host.begin());
		thrust::host_vector<J_row> J_host(n_total);
		thrust::copy(J->begin(), J->begin() + n_total, J_host.begin());

		for (size_t i = 0; i < NUM_THETAS; i++) {
			double result = 0;
			for (size_t k = 0; k < n_total; k++) {
				J_row a = J_host[k];
				result = result + (double)a.data[i] * (double)F_host[k];
			}
			eigen_JtF[i] = result;
		}
		for (size_t i = 0; i < NUM_THETAS; i++) {
			for (size_t j = 0; j < NUM_THETAS; j++) {
				double result = 0;
				for (size_t k = 0; k < n_total; k++) {
					J_row a = J_host[k];
					result = result + (double)a.data[i] * (double)a.data[j];
				}
				eigen_JtJ[i * NUM_THETAS + j] = result;
			}
		}
	}

	/// Only need evaluate metric on the last iteration
	if (eval_metric) {
		thrust::device_vector<float> f_pull(n_pull);
		thrust::transform(F->begin() + n_push, F->begin() + n_push + n_pull, f_pull.begin(), absolute_value());
		pull_error = thrust::reduce(f_pull.begin(), f_pull.end());
		pull_error = pull_error / n_pull;
		// pull_num = n_pull;
		//std::cout << pull_error << std::endl;

		thrust::device_vector<float> f_push(n_push);
		thrust::transform(F->begin(), F->begin() + n_push, f_push.begin(), absolute_value());
		push_error = thrust::reduce(f_push.begin(), f_push.end());
		push_error = push_error / n_push;
		// push_num = n_push;
	}

	//Write the correspondences	
	if (store_corresps) {
		std::ofstream output_file;
		std::string data_path = "...";
		thrust::host_vector<float> output(n_pull * factor);

		thrust::copy(hmodel_correspondences->begin(), hmodel_correspondences->begin() + n_pull * factor, output.begin());
		output_file.open(data_path + "corresp-" + std::to_string(id) + ".txt");
		for (size_t i = 0; i < n_pull * factor; i++) {
			output_file << output[i] << " ";
		}
		output_file.close();

	}
	return;
}


void kernel3(float* eigen_JtJ, float* eigen_JtF, bool reweight, int num_sensor_points) {

	//CUDA_TIMED_BLOCK(timer, "indexing constraints");

	int n_pull, n_push;

//	if (!settings->fit2D_enable) 
	{
		n_pull = 1 * num_sensor_points; // point to plane
		n_push = 0;
	}

	int n_total = n_pull + n_push;

	// CUDA_TIMED_BLOCK(timer,"memory resize + zero (J+e)")
	{
		const J_row zeros = {};
		thrust::fill(J2->begin(), J2->begin() + n_total, zeros);
		thrust::fill(F2->begin(), F2->begin() + n_total, 0.0f);

		if (n_total == 0) return;
	}
	J_row* J_push = thrust::raw_pointer_cast(J2->data());
	J_row* J_pull = J_push + n_push;

	float* F_push = thrust::raw_pointer_cast(F2->data());
	float* F_pull = F_push + n_push;

	ComputeJacobianDataPointCloudMultiCamera functor_data_model(J_pull, F_pull, reweight);


	//CUDA_TIMED_BLOCK(timer,"Assemble Jacobian")
	{
		if (settings->fit3D_enable)
			thrust::for_each(_point_index->begin(), _point_index->begin() + num_sensor_points, functor_data_model);
	}

	// CUDA_TIMED_BLOCK(timer, "Jt*J and Jt*e + CPU Transfer")
	{
		CublasHelper::outer_product_J(*J2, *JtJ2, n_total, NUM_THETAS);
		CublasHelper::vector_product_J(*J2, *F2, *JtF2, n_total, NUM_THETAS);
		thrust::copy(JtF2->begin(), JtF2->end(), eigen_JtF);
		thrust::copy(JtJ2->begin(), JtJ2->end(), eigen_JtJ);
	}

	return;
}

void kernel_interaction_joints(float* eigen_JtJ, float* eigen_JtF, int num_points,bool store_result, int frame_idx) {

	int n_total = num_points;
	if (n_total < 60) return;
	printf("interaction points number:%d\n", num_points);
	// CUDA_TIMED_BLOCK(timer,"memory resize + zero (J+e)")
	{
		const J_row zeros = {};
		thrust::fill(J2->begin(), J2->begin() + n_total, zeros);
		thrust::fill(F2->begin(), F2->begin() + n_total, 0.0f);
	}

	J_row* J_interaction = thrust::raw_pointer_cast(J2->data());

	float* F_interaction = thrust::raw_pointer_cast(F2->data());

	ComputeJacobianInteractionJoints functor_interaction_model(J_interaction, F_interaction);

	thrust::sequence(interaction_indices->begin(), interaction_indices->begin() + n_total);

	//CUDA_TIMED_BLOCK(timer,"Assemble Jacobian")
	{
		thrust::for_each(interaction_indices->begin(), interaction_indices->begin() + n_total, functor_interaction_model);
	}

	// CUDA_TIMED_BLOCK(timer, "Jt*J and Jt*e + CPU Transfer")
	{
		CublasHelper::outer_product_J(*J2, *JtJ2, n_total, NUM_THETAS);
		CublasHelper::vector_product_J(*J2, *F2, *JtF2, n_total, NUM_THETAS);

		/*thrust::host_vector<float> output(n_total);
		thrust::copy(F2->begin(), F2->begin() + n_total, output.begin());
		thrust::host_vector<J_row> Jacobian_out(n_total);
		thrust::copy(J2->begin(), J2->begin() + n_total, Jacobian_out.begin());*/

		thrust::copy(JtF2->begin(), JtF2->end(), eigen_JtF);
		thrust::copy(JtJ2->begin(), JtJ2->end(), eigen_JtJ);
	}

	//Write the value of F2 to host	
	/*if (store_result) {
		std::ofstream output_file;
		std::string data_path = "";
		thrust::host_vector<float> output(n_total);

		thrust::copy(F2->begin(), F2->begin() + n_total, output.begin());
		output_file.open(data_path + "F_interaction" + std::to_string(frame_idx) + ".txt");
		output_file << "total number:" << n_total << std::endl;
		for (size_t i = 0; i < n_total; i++) {
			output_file <<output[i] << " ";
		}
		output_file.close();
	}*/

	return;
}

void kernel_interaction_blocks(float* eigen_JtJ, float* eigen_JtF, int num_points, bool store_result, int frame_idx) {

	int n_total = num_points;
	if (n_total < 10) return;
	//	printf("n_total:%d\n", n_total);

	// CUDA_TIMED_BLOCK(timer,"memory resize + zero (J+e)")
	{
		const J_row zeros = {};
		thrust::fill(J2->begin(), J2->begin() + n_total, zeros);
		thrust::fill(F2->begin(), F2->begin() + n_total, 0.0f);
	}

	J_row* J_interaction = thrust::raw_pointer_cast(J2->data());

	float* F_interaction = thrust::raw_pointer_cast(F2->data());

	ComputeJacobianInteractionBlocks functor_interaction_model(J_interaction, F_interaction);

	thrust::sequence(interaction_indices->begin(), interaction_indices->begin() + n_total);

	//CUDA_TIMED_BLOCK(timer,"Assemble Jacobian")
	{
		thrust::for_each(interaction_indices->begin(), interaction_indices->begin() + n_total, functor_interaction_model);
	}

	// CUDA_TIMED_BLOCK(timer, "Jt*J and Jt*e + CPU Transfer")
	{
		CublasHelper::outer_product_J(*J2, *JtJ2, n_total, NUM_THETAS);
		CublasHelper::vector_product_J(*J2, *F2, *JtF2, n_total, NUM_THETAS);
		thrust::copy(JtF2->begin(), JtF2->end(), eigen_JtF);
		thrust::copy(JtJ2->begin(), JtJ2->end(), eigen_JtJ);
	}

	//Write the value of F2 to host	
	/*if (store_result) {
	std::ofstream output_file;
	std::string data_path = "";
	thrust::host_vector<float> output(n_total);

	thrust::copy(F2->begin(), F2->begin() + n_total, output.begin());
	output_file.open(data_path + "F_interaction" + std::to_string(frame_idx) + ".txt");
	output_file << "total number:" << n_total << std::endl;
	for (size_t i = 0; i < n_total; i++) {
	output_file <<output[i] << " ";
	}
	output_file.close();
	}*/

	return;
}

void kernel_get_conf(int* conf)
{
	if (!conf)
		return;
	thrust::host_vector<int> host_conf = *device_confidence;
	for (int i = 0; i < host_conf.size(); i++)
		conf[i] = host_conf[i];
}
